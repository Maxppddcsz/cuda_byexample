
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void hello_from_gpu(){

    printf("Hello World from block %d and thread (%d-%d)\n",blockIdx.x,threadIdx.x,threadIdx.y);
}
int main(){
    const dim3 block_size(2,4);
    hello_from_gpu<<<1,block_size>>>();
    /*
    一块GPU中有很多计算核心，可以支持很多线程（thread）。主机在调用一个核函数时
    必须指明需要在设备中指派多少个线程，否则设备不知道如何工作。三括号中的数就是
    用来指明核函数中的线程数目及排列情况的。
    核函数中的线程常组织为若干线程块（threadblock）:三括号中的第一个数字可以看作
    线程块的个数，第二个数字可以看作每个线程块中的线程数。一个核函数的全部线程块
    构成一个网格（grid），而线程块的个数就记为网格大小（grid size）。每个线程块中
    含有同样数目的线程，该数目称为线程块大小（blocksize）。所以核函数中总的线程数
    就等于网格大小乘以线程块大小，而三括号中的两个数字分别为网格大小和线程块大小，
    即＜＜＜网格大小，线程块大小＞＞＞。所以,在上述程序中，主机只指派了设备的一个线程，
    网格大小和线程块大小都是1,即1×1＝1
    */
    hipDeviceSynchronize();
    /*
    调用输出函数时，输出流是先存放在缓冲区的，而缓冲区不会自动刷新。只有程序遇到某种同步
    操作时缓冲区才会刷新。函数 cudaDeviceSynchronize()的作用是同步主机与设备，所以能够
    促使缓冲区刷新。
    */
    return 0;
}